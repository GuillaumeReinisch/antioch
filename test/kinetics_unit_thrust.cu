#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------bl-
//--------------------------------------------------------------------------
//
// Antioch - A Gas Dynamics Thermochemistry Library
//
// Copyright (C) 2013 The PECOS Development Team
//
// This library is free software; you can redistribute it and/or
// modify it under the terms of the Version 2.1 GNU Lesser General
// Public License as published by the Free Software Foundation.
//
// This library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
// Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public
// License along with this library; if not, write to the Free Software
// Foundation, Inc. 51 Franklin Street, Fifth Floor,
// Boston, MA  02110-1301  USA
//
//-----------------------------------------------------------------------el-
//
// $Id$
//
//--------------------------------------------------------------------------
//--------------------------------------------------------------------------

// Antioch
#include "antioch/vector_utils.h"

#include "antioch/antioch_asserts.h"
#include "antioch/chemical_species.h"
#include "antioch/chemical_mixture.h"
#include "antioch/reaction_set.h"
#include "antioch/read_reaction_set_data_xml.h"
#include "antioch/cea_thermo.h"
#include "antioch/kinetics_evaluator.h"

// Thrust
#include "thrust/for_each.h"
#include "thrust/device_vector.h"
#include "thrust/iterator/zip_iterator.h"

// C++
#include <limits>
#include <iostream>
#include <string>
#include <vector>

template <typename Scalar>
struct antioch_functor
{
  antioch_functor(const std::vector<std::string> &species_str_list,
		  const std::string& reaction_data_filename) :
    chem_mixture(species_str_list),
    reaction_set(chem_mixture),
    thermo(chem_mixture),
    kinetics(reaction_set, 0)
  { 
    Antioch::read_reaction_set_data_xml<Scalar>
      (reaction_data_filename, false, reaction_set);

    P = 1.0e5;
    Y.resize(species_str_list.size(),1.0/species_str_list.size());
    R_mix = chem_mixture.R(Y);
  }

  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    const Scalar& T = thrust::get<0>(t);
    const Scalar rho = P / (R_mix * T);

    const short n_species = Y.size();
    std::vector<Scalar> molar_densities(n_species,0.0),
		        h_RT_minus_s_R (n_species),
			omega_dot(n_species);

    chem_mixture.molar_densities(rho,Y,molar_densities);

    typedef typename Antioch::CEAThermodynamics<Scalar>::template Cache<Scalar> Cache;

    thermo.h_RT_minus_s_R(Cache(T),h_RT_minus_s_R);

    kinetics.compute_mass_sources( T, rho, R_mix, Y, molar_densities, h_RT_minus_s_R, omega_dot );
  }

  // Make these first three shared?
  Antioch::ChemicalMixture<Scalar>   chem_mixture;
  Antioch::ReactionSet<Scalar>       reaction_set;
  Antioch::CEAThermodynamics<Scalar> thermo;

  // Need one of these per object
  Antioch::KineticsEvaluator<Scalar> kinetics;

  // Matching the other unit tests
  Scalar P;
  std::vector<Scalar> Y;
  Scalar R_mix;
};

template <typename Scalar>
int tester_N2N(const std::string& input_name)
{
  using std::abs;

  std::vector<std::string> species_str_list;
  const unsigned int n_species = 2;
  species_str_list.reserve(n_species);
  species_str_list.push_back( "N2" );
  species_str_list.push_back( "N" );

  int return_flag = 0;

/*
  for( unsigned int i = 0; i < n_T_samples; i++ )
    {
      // Omega dot had better sum to 0.0
      Scalar sum = 0;
      for( unsigned int s = 0; s < n_species; s++ )
	{
	  sum += omega_dot[s];
	}
      const Scalar sum_tol = std::numeric_limits<Scalar>::epsilon() * 1.0e6; // 1.0e-10;
      if( abs( sum ) > sum_tol )
	{
	  return_flag = 1;
	  std::cerr << "Error: omega_dot did not sum to 0.0." << std::endl
		    << std::scientific << std::setprecision(16)
		    << "T = " << T << std::endl
		    << "sum = " << sum << ", sum_tol = " << sum_tol << std::endl;
	  for( unsigned int s = 0; s < n_species; s++ )
	    {
	      std::cerr << std::scientific << std::setprecision(16)
			<< "omega_dot(" << chem_mixture.chemical_species()[s]->species() << ") = "
			<< omega_dot[s] << std::endl;
	    }
	  std::cout << std::endl << std::endl;
	}
    }
*/
  
  return return_flag;
}


template <typename Scalar>
int tester(const std::string& input_name)
{
  using std::abs;

  std::vector<std::string> species_str_list;
  const unsigned int n_species = 5;
  species_str_list.reserve(n_species);
  species_str_list.push_back( "N2" );
  species_str_list.push_back( "O2" );
  species_str_list.push_back( "N" );
  species_str_list.push_back( "O" );
  species_str_list.push_back( "NO" );

  int return_flag = 0;

/*
  for( unsigned int i = 0; i < n_T_samples; i++ )
    {
      // Omega dot had better sum to 0.0
      Scalar sum = 0;
      for( unsigned int s = 0; s < n_species; s++ )
	{
	  sum += omega_dot[s];
	}
      const Scalar sum_tol = std::numeric_limits<Scalar>::epsilon() * 2.7e6; // 1.6e-10;
      if( abs( sum ) > sum_tol )
	{
	  return_flag = 1;
	  std::cerr << "Error: omega_dot did not sum to 0.0." << std::endl
		    << std::scientific << std::setprecision(16)
		    << "T = " << T << std::endl
		    << "sum = " << sum << ", sum_tol = " << sum_tol << std::endl;
	  for( unsigned int s = 0; s < n_species; s++ )
	    {
	      std::cerr << std::scientific << std::setprecision(16)
			<< "omega_dot(" << chem_mixture.chemical_species()[s]->species() << ") = "
			<< omega_dot[s] << std::endl;
	    }
	  std::cout << std::endl << std::endl;
	}
    }
*/
  
  return return_flag;
}


int main(int argc, char* argv[])
{
  // Check command line count.
  if( argc < 2 )
    {
      // TODO: Need more consistent error handling.
      std::cerr << "Error: Must specify reaction set XML input file." << std::endl;
      antioch_error();
    }

  int return_flag=0;

  return_flag += (tester<double>(std::string(argv[1])) ||
                  tester<long double>(std::string(argv[1])) ||
                  tester<float>(std::string(argv[1])));

  return_flag += (tester_N2N<double>(std::string(argv[1])) ||
                  tester_N2N<long double>(std::string(argv[1])) ||
                  tester_N2N<float>(std::string(argv[1])));

  return return_flag;
}

